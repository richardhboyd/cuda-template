#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvtx3/nvtx3.hpp> // Include the NVTX3 header
#include "myKernel.cuh"

__device__ int cube (int x)
{
  int y;
  asm(".reg .u32 t1;\n\t"              // temp reg t1
      " mul.lo.u32 t1, %1, %1;\n\t"    // t1 = x * x
      " mul.lo.u32 %0, t1, %1;"        // y = t1 * x
      : "=r"(y) : "r" (x));
  return y;
}

__device__ int cpp_cube(int x)
{
  int y = x*x*x;
  return y;
}

__global__ void asm_kernel(int input, float *out) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int x = cube(idx);
    out[idx] = x;
}

__global__ void simple_kernel(int input, float *out) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int y = cpp_cube(idx);
    out[idx] = y;
}

void call_cuda_kernel(int input) {
    float *A, *B, *d_A, *d_B;
    int A_size = 32;
    int B_size = 32;
    A = (float*)malloc(A_size*sizeof(int));
    B = (float*)malloc(B_size*sizeof(int));
    hipMalloc(&d_A, A_size*sizeof(int));
    hipMalloc(&d_B, B_size*sizeof(int));

    nvtxRangePushA("asm cube"); 
    asm_kernel<<<2, 32>>>(input, d_A);
    hipDeviceSynchronize();
    hipMemcpy(A, d_A, A_size*sizeof(int), hipMemcpyDeviceToHost);
    nvtxRangePop();

    nvtxRangePushA("cpp cube"); 
    simple_kernel<<<2, 32>>>(input, d_B);
    hipDeviceSynchronize();
    hipMemcpy(B, d_B, B_size*sizeof(int), hipMemcpyDeviceToHost);
    nvtxRangePop();
}