#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "myKernel.cuh"

__global__ void simple_kernel() {
    printf("Hello from CUDA kernel!\n");
}

void call_cuda_kernel() {
    simple_kernel<<<1, 32>>>();
    hipDeviceSynchronize(); // Ensure kernel completes before returning
}